
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify=expected,onhost %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only -fcuda-is-device -verify=expected,ondevice %s

template <bool C, class T = void> struct my_enable_if {};

template <class T> struct my_enable_if<true, T> {
  typedef T type;
};

__attribute__((host, device)) void use(int x);

__attribute__((device)) constexpr int OverloadFunHostDepr(void) { return 1; }
__attribute__((host, deprecated("Host variant"))) constexpr int OverloadFunHostDepr(void) { return 1; } // expected-note 0+ {{has been explicitly marked deprecated here}}


__attribute__((device, deprecated("Device variant"))) constexpr int OverloadFunDeviceDepr(void) { return 1; } // expected-note 0+ {{has been explicitly marked deprecated here}}
__attribute__((host)) constexpr int OverloadFunDeviceDepr(void) { return 1; }


template<typename T>
__attribute__((device)) constexpr T TemplateOverloadFun(void) { return 1; }

template<typename T>
__attribute__((host, deprecated("Host variant"))) constexpr T TemplateOverloadFun(void) { return 1; } // expected-note 0+ {{has been explicitly marked deprecated here}}


__attribute__((device, deprecated)) constexpr int // expected-note 0+ {{has been explicitly marked deprecated here}}
DeviceOnlyFunDeprecated(void) { return 1; }

__attribute__((host, deprecated)) constexpr int // expected-note 0+ {{has been explicitly marked deprecated here}}
HostOnlyFunDeprecated(void) { return 1; }

class FunSelector {
public:
  template<int X> __attribute__((device))
  auto devicefun(void) -> typename my_enable_if<(X == OverloadFunHostDepr()), int>::type {
    return 1;
  }

  template<int X> __attribute__((device))
  auto devicefun(void) -> typename my_enable_if<(X != OverloadFunHostDepr()), int>::type {
      return 0;
  }

  template<int X> __attribute__((device))
  auto devicefun_wrong(void) -> typename my_enable_if<(X == OverloadFunDeviceDepr()), int>::type { // ondevice-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}
    return 1;
  }

  template<int X> __attribute__((device))
  auto devicefun_wrong(void) -> typename my_enable_if<(X != OverloadFunDeviceDepr()), int>::type { // ondevice-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}
      return 0;
  }

  template<int X> __attribute__((host))
  auto hostfun(void) -> typename my_enable_if<(X == OverloadFunDeviceDepr()), int>::type {
    return 1;
  }

  template<int X> __attribute__((host))
  auto hostfun(void) -> typename my_enable_if<(X != OverloadFunDeviceDepr()), int>::type {
      return 0;
  }

  template<int X> __attribute__((host))
  auto hostfun_wrong(void) -> typename my_enable_if<(X == OverloadFunHostDepr()), int>::type { // onhost-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
    return 1;
  }

  template<int X> __attribute__((host))
  auto hostfun_wrong(void) -> typename my_enable_if<(X != OverloadFunHostDepr()), int>::type { // onhost-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
      return 0;
  }
};


// These should not be diagnosed since the device overload of
// OverloadFunHostDepr is not deprecated:
__attribute__((device)) my_enable_if<(OverloadFunHostDepr() > 0), int>::type
DeviceUserOverloadFunHostDepr1(void) { return 2; }

my_enable_if<(OverloadFunHostDepr() > 0), int>::type __attribute__((device))
DeviceUserOverloadFunHostDepr2(void) { return 2; }

__attribute__((device))
my_enable_if<(OverloadFunHostDepr() > 0), int>::type constexpr
DeviceUserOverloadFunHostDeprConstexpr(void) { return 2; }


// Analogously for OverloadFunDeviceDepr:
__attribute__((host)) my_enable_if<(OverloadFunDeviceDepr() > 0), int>::type
DeviceUserOverloadFunDeviceDepr1(void) { return 2; }

my_enable_if<(OverloadFunDeviceDepr() > 0), int>::type __attribute__((host))
DeviceUserOverloadFunDeviceDepr2(void) { return 2; }

__attribute__((host))
my_enable_if<(OverloadFunDeviceDepr() > 0), int>::type constexpr
DeviceUserOverloadFunDeviceDeprConstexpr(void) { return 2; }


// Actual uses of the deprecated overloads should be diagnosed:
__attribute__((host, device)) my_enable_if<(OverloadFunHostDepr() > 0), int>::type // onhost-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
HostDeviceUserOverloadFunHostDepr(void) { return 3; }

__attribute__((host)) my_enable_if<(OverloadFunHostDepr() > 0), int>::type constexpr // onhost-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
HostUserOverloadFunHostDeprConstexpr(void) { return 3; }

__attribute__((device)) my_enable_if<(OverloadFunDeviceDepr() > 0), int>::type constexpr // ondevice-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}
HostUserOverloadFunDeviceDeprConstexpr(void) { return 3; }


// Making the offending decl a template shouldn't change anything:
__attribute__((host)) my_enable_if<(TemplateOverloadFun<int>() > 0), int>::type // onhost-warning {{'TemplateOverloadFun<int>' is deprecated: Host variant}}
HostUserTemplateOverloadFun(void) { return 3; }

__attribute__((device)) my_enable_if<(TemplateOverloadFun<int>() > 0), int>::type
DeviceUserTemplateOverloadFun(void) { return 3; }


// If the constexpr function is actually called from the mismatched context, diagnostics should be issued:
__attribute__((host))
my_enable_if<(DeviceOnlyFunDeprecated() > 0), int>::type constexpr // onhost-warning {{'DeviceOnlyFunDeprecated' is deprecated}}
HostUserDeviceOnlyFunDeprecated(void) { return 3; }

__attribute__((device))
my_enable_if<(HostOnlyFunDeprecated() > 0), int>::type constexpr // ondevice-warning {{'HostOnlyFunDeprecated' is deprecated}}
DeviceUserHostOnlyFunDeprecated(void) { return 3; }

// Diagnostics for uses in function bodies should work as expected:
__attribute__((device, deprecated)) constexpr int DeviceVarConstDepr = 1; // expected-note 0+ {{has been explicitly marked deprecated here}}

__attribute__((host)) void HostUser(void) {
  use(DeviceVarConstDepr); // expected-warning {{'DeviceVarConstDepr' is deprecated}}
  use(HostOnlyFunDeprecated()); // expected-warning {{'HostOnlyFunDeprecated' is deprecated}}
  use(OverloadFunHostDepr()); // expected-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
  use(TemplateOverloadFun<int>()); // expected-warning {{'TemplateOverloadFun<int>' is deprecated: Host variant}}

  use(OverloadFunDeviceDepr());
}

__attribute__((device)) void DeviceUser(void) {
  use(DeviceVarConstDepr); // expected-warning {{'DeviceVarConstDepr' is deprecated}}
  use(DeviceOnlyFunDeprecated()); // expected-warning {{'DeviceOnlyFunDeprecated' is deprecated}}
  use(OverloadFunDeviceDepr()); // expected-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}

  use(OverloadFunHostDepr());
  use(TemplateOverloadFun<int>());
}
